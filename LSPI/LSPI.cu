#include "hip/hip_runtime.h"
/**
 * LSPI_CPP.cpp : Defines the entry point for the console application.
 *
 * Executes a series of tests of different Agents trying to solve the inverted pendulum problem.
 * Tracks the performance of each type of agent across multiple trials and prints the results.
 */

#include "stdafx.h"
#include "Agent.h"
#include "LspiAgent.h"
#include "NoopAgent.h"
#include "CleverAgent.h"
#include <vector>
#include <array>
#include <time.h>
#include "Pendulum.h"
#include <Windows.h>
#include <conio.h>
#include "TestBlas.h"
#include "hip/hip_math_constants.h"
#include <thrust\host_vector.h>
#include "sample.h"
#include <string.h>
#include <stdlib.h>
#include <fstream>


using namespace std;

#define DT_CONST 0.1f
#define NUM_TRIALS 10000
#define NUM_SAMPLE_TRIALS 10000
#define DISCOUNT 0.95f

#define TEST_FIRST
#define USE_FILE // If defined, samples will be pulled from a file titled samples.txt instead of from randomly generated input

/**
 * Calculates the time between the two clock events. Currently is not working as expected.
 */
double diffclock(clock_t clock1,clock_t clock2)
{
	double diffticks= clock2-clock1;
	double diffms=(diffticks*10)/CLOCKS_PER_SEC;
	return diffms;
}

inline void getSamplesFromFile(string filename, thrust::host_vector<sample>& samples)
{
	ifstream file(filename);
	string value;
	thrust::host_vector<sample>::iterator it = samples.end(); 
	while(file.good())
	{
		sample s;
		getline(file, value, ',');
		s.angle = (float)atof(value.c_str());

		getline(file, value, ',');
		s.angular_velocity = (float)atof(value.c_str());

		getline(file, value, ',');
		s.action = atoi(value.c_str());

		getline(file, value, ',');
		s.reward = atoi(value.c_str());

		getline(file, value, ',');
		s.final_angle = (float)atof(value.c_str());

		getline(file, value, ',');
		s.final_angular_velocity = (float)atof(value.c_str());

		getline(file, value, '\n');
		s.terminal = atoi(value.c_str());

		samples.insert(it, s);
		it = samples.end();
	}
}

int _tmain(int, _TCHAR*)
{
#ifdef TEST_FIRST
	if(!TestBlas::run_tests())
		getch();
#endif

	hipblasStatus_t stat = hipblasCreate(&blas::handle);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("CUBLAS Init Failure.");
		return -1;
	}

	printf("%d", NUM_SAMPLE_TRIALS);
	srand((unsigned int)time(NULL));
	int random_agent_life = 0;
	int clever_agent_life = 0;
	int noop_agent_life = 0;
	int lspi_agent_life = 0;

	thrust::host_vector<sample> samples;

#if defined(USE_FILE)
	getSamplesFromFile("D:\\Users\\Ithiel\\Documents\\College\\Thesis\\samples.txt", samples);
#else
	thrust::host_vector<sample>::iterator it = samples.end(); 
	for(int i = 0; i < NUM_SAMPLE_TRIALS; i++)
	{
		Pendulum pen;
		Agent agent;
		while(!pen.isHorizontal())
		{
			// Track the random agent's samples
			sample s;
			s.angle = pen.x;
			s.angular_velocity = pen.v;

			int action = agent.getAction(pen.x, pen.v);
			pen.update(DT_CONST, action);
			int reward = pen.isHorizontal() ? -1 : 0;
			agent.notify(reward);

			s.action = action;
			s.reward = reward;
			s.final_angle = pen.x;
			s.final_angular_velocity = pen.v;

			if (reward < 0)
			{
				s.terminal = 1;
			}
			else
			{
				s.terminal = 0;
			}
			samples.insert(it, s);
			it = samples.end();
		}
	}
#endif

	clock_t start = clock();
//	LspiAgent<host_vector<float>> lspi_agent(samples, DISCOUNT);
	LspiAgent<device_vector<float>> lspi_agent(samples, DISCOUNT); 
	clock_t end = clock();
	printf("Single-threaded: %f\n", diffclock(start, end));

	for(int i = 0; i < NUM_TRIALS; i++)
	{
		{
			Pendulum pen;
			Agent agent;
			while(!pen.isHorizontal())
			{
				int action = agent.getAction(pen.x, pen.v);
				pen.update(DT_CONST, action);
				random_agent_life += 1;
			}
		}

		{
			Pendulum pen;
			CleverAgent agent;
			while(!pen.isHorizontal())
			{
				int action = agent.getAction(pen.x, pen.v);
				pen.update(DT_CONST, action);
				clever_agent_life += 1;
			}
		}

		{
			Pendulum pen;
			NoopAgent agent;
			while(!pen.isHorizontal())
			{
				int action = agent.getAction(pen.x, pen.v);
				pen.update(DT_CONST, action);
				noop_agent_life += 1;
			}
		}

		{
			Pendulum pen;
			int temp_life = 0;
			while(!pen.isHorizontal() && temp_life < 3000)
			{
				int action = lspi_agent.getAction(pen.x, pen.v);
				pen.update(DT_CONST, action);
				lspi_agent_life += 1;
				temp_life += 1;
			}
		}
	}

	printf("\nSummary:\n");
	printf("Random Agent: %f\n", (double)(random_agent_life*DT_CONST)/NUM_TRIALS);
	printf("Clever Agent: %f\n", (double)(clever_agent_life*DT_CONST)/NUM_TRIALS);
	printf("No-Op Agent: %f\n", (double)(noop_agent_life*DT_CONST)/NUM_TRIALS);
	printf("LSPI Agent: %f\n", (double)(lspi_agent_life*DT_CONST)/NUM_TRIALS);

	// Wait so we can get the results
	getch();

	return 0;
}