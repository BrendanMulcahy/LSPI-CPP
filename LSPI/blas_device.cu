/**
 * Provides a wrapper around cublas for computation on the GPU.
 */

#include "stdafx.h"
#include "blas.h"

hipblasHandle_t blas::handle;

/**
* Computes C = alpha*A*B + beta*C
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemm(const Matrix<device_vector<float>>& A, const Matrix<device_vector<float>>& B, Matrix<device_vector<float>>& C, float alpha, float beta)
{
	hipblasStatus_t status = hipblasSgemm(blas::handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C.rows, C.cols, A.cols, &alpha, raw_pointer_cast(A.vector.data()), 
										A.rows, raw_pointer_cast(B.vector.data()), B.rows, &beta, raw_pointer_cast(C.vector.data()), C.rows);

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes C = alpha*A*B
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemm(const Matrix<device_vector<float>>& A, const Matrix<device_vector<float>>& B, Matrix<device_vector<float>>& C, float alpha)
{
	return blas::gemm(A, B, C, alpha, 0.0);
}

/**
* Computes C = A*B
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemm(const Matrix<device_vector<float>>& A, const Matrix<device_vector<float>>& B, Matrix<device_vector<float>>& C)
{
	return blas::gemm(A, B, C, 1.0, 0.0);
}

/**
* Computes A = alpha*A
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemm(Matrix<device_vector<float>>& A, float alpha)
{
	return blas::gemm(A, A, A, 0.0, 1.0);
}

/**
* Computes x = alpha*x. 
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::scal(device_vector<float>& x, float alpha)
{
	hipblasStatus_t status = hipblasSscal(blas::handle, x.size(), &alpha, raw_pointer_cast(x.data()), 1);

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes result = x dot y
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::dot(const device_vector<float>& x, const device_vector<float>& y, float& result)
{
	hipblasStatus_t status = hipblasSdot(blas::handle, x.size(), raw_pointer_cast(x.data()), 1, raw_pointer_cast(y.data()), 1, &result);

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}
	
/**
* Computes y = alpha*A*x + beta*y. For alpha*x*A set transpose to true.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemv(const Matrix<device_vector<float>>& A, const device_vector<float>& x, device_vector<float>& y, float alpha, float beta, bool transpose)
{
	hipblasStatus_t status;
	
	if(transpose)
	{
		status = hipblasSgemv(blas::handle, HIPBLAS_OP_T, A.rows, A.cols, &alpha, raw_pointer_cast(A.vector.data()), A.rows, raw_pointer_cast(x.data()),
						     1, &beta, raw_pointer_cast(y.data()), 1);
	}
	else
	{
		status = hipblasSgemv(blas::handle, HIPBLAS_OP_N, A.rows, A.cols, &alpha, raw_pointer_cast(A.vector.data()), A.rows, raw_pointer_cast(x.data()),
							 1, &beta, raw_pointer_cast(y.data()), 1);
	}

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes y = alpha*A*x. For alpha*x*A set transpose to true.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemv(const Matrix<device_vector<float>>& A, const device_vector<float>& x, device_vector<float>& y, float alpha, bool transpose)
{
	return blas::gemv(A, x, y, alpha, 0.0, transpose);
}

/**
* Computes y = alpha*A*x. For alpha*x*A set tranpose to true.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::gemv(const Matrix<device_vector<float>>& A, const device_vector<float>& x, device_vector<float>& y, bool transpose)
{
	return blas::gemv(A, x, y, 1.0, 0.0, transpose);
}

/**
* Computes C = alpha*A + beta*B.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::geam(const Matrix<device_vector<float>>& A, const Matrix<device_vector<float>>& B, Matrix<device_vector<float>>& C, float alpha, float beta)
{
	hipblasStatus_t status = hipblasSgeam(blas::handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A.rows, A.cols, &alpha, raw_pointer_cast(A.vector.data()),
									    A.rows, &beta, raw_pointer_cast(B.vector.data()), B.rows, raw_pointer_cast(C.vector.data()), C.rows);

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes C = A + B.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::geam(const Matrix<device_vector<float>>& A, const Matrix<device_vector<float>>& B, Matrix<device_vector<float>>& C)
{
	return blas::geam(A, B, C, 1.0, 1.0);
}

/**
* Computes y = alpha*x + y
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::axpy(const device_vector<float>& x, device_vector<float>& y, float alpha)
{
	hipblasStatus_t status = hipblasSaxpy(blas::handle, x.size(), &alpha, raw_pointer_cast(x.data()), 1, raw_pointer_cast(y.data()), 1);
	
	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes y = x + y
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::axpy(const device_vector<float>& x, device_vector<float>& y)
{
	return blas::axpy(x, y, 1.0);
}

/**
* Computes A = alpha*x*y.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::ger(const device_vector<float>& x, const device_vector<float>& y, Matrix<device_vector<float>>& A, float alpha)
{
	hipblasStatus_t status = hipblasSger(blas::handle, x.size(), y.size(), &alpha, raw_pointer_cast(x.data()), 1, raw_pointer_cast(y.data()), 1,
									   raw_pointer_cast(A.vector.data()), A.rows);

	if(status == HIPBLAS_STATUS_SUCCESS)
	{
		return 0;
	}
	else
	{
		return -1;
	}
}

/**
* Computes A = x*y.
* Returns 0 if the operation was successful, an error code otherwise
*/
int blas::ger(const device_vector<float>& x, const device_vector<float>& y, Matrix<device_vector<float>>& A)
{
	return blas::ger(x, y, A, 1.0);
}